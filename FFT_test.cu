
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include <unistd.h>



int main(int argc, char** argv) {

  int sz = 1024;
  int nGPUs = 2, whichGPUs[16];
  whichGPUs[0] = 0; whichGPUs[1] = 1;
  if (argc > 1) {
    if ( ( strncmp(argv[1], "--help", 6) == 0) ||
         ( strncmp(argv[1], "-h", 2) == 0) ) {
      std::cout << "Usage:\n     FFT_test <field dimension> <num GPUs>" <<std::endl;
      return 0;
    } 
    if ( argv[1][strlen(argv[1])-1] == 'k') sz = atoi(argv[1])*1024;
    else sz = atoi(argv[1]);
  }
  int nx=sz, ny=sz;

  if (argc > 2) {
    nGPUs = atoi(argv[2]);
    if (nGPUs > 16) {
       std::cout << "More than 16 GPUs not supported" << std::endl;
       exit(1);
    }
    for (size_t qq=0;qq<nGPUs;qq++) whichGPUs[qq]=qq;
  }
  std::cout << "Transform " << ny << "x" << nx << " image "
            << "with " << nGPUs << " gpus." << std::endl;

// Timers
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEvent_t mem_start, mem_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&mem_start);
    hipEventCreate(&mem_stop);

    hipfftHandle plan_input; hipfftResult result; 

//
// Initialize FFT input data
    size_t worksize[16];
    hipfftComplex *host_data_input, *host_data_output;
    unsigned long size_of_data = sizeof(hipfftComplex) * nx * ny;
    host_data_input = (hipfftComplex *)malloc(size_of_data);
    if (host_data_input == NULL) { printf ("malloc (%ul) failed\n", size_of_data); return 0; }
    host_data_output = (hipfftComplex *)malloc(size_of_data);
    if (host_data_output == NULL) { printf ("malloc (%ul, output) failed\n", size_of_data); return 0; }
    //initialize_2d_data (nx, ny, host_data_input, host_data_output);

  if (nGPUs > 1) {
// Demonstrate how to use CUFFT to perform 2-d FFTs using multiple GPUs
//
// cufftCreate() - Create an empty plan
    result = hipfftCreate(&plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Create failed\n"); return 0; }
//
// cufftXtSetGPUs() - Define which GPUs to use
    result = hipfftXtSetGPUs (plan_input, nGPUs, whichGPUs);
    if (result != HIPFFT_SUCCESS) { printf ("*XtSetGPUs failed\n"); return 0; }
//
// cufftMakePlan2d() - Create the plan
    result = hipfftMakePlan2d (plan_input, ny, nx, HIPFFT_C2C, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed\n"); return 0; }
//
// cufftXtMalloc() - Malloc data on multiple GPUs
    hipLibXtDesc *device_data_input;
    result = hipfftXtMalloc (plan_input, &device_data_input,
        HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); return 0; }
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
//
// cufftXtMemcpy() - Copy data from host to multiple GPUs
    hipSetDevice(0); hipEventRecord(mem_start);
    result = hipfftXtMemcpy (plan_input, device_data_input,
        host_data_input, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
//
// cufftXtExecDescriptorC2C() - Execute FFT on multiple GPUs
    hipSetDevice(0); hipEventRecord(start);
    result = hipfftXtExecDescriptorC2C (plan_input, device_data_input,
        device_data_input, HIPFFT_FORWARD);
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
    hipSetDevice(0); hipEventRecord(stop);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExec* failed\n"); return 0; }
//
// cufftXtMemcpy() - Copy data from multiple GPUs to host
    result = hipfftXtMemcpy (plan_input, host_data_output,
        device_data_input, HIPFFT_COPY_DEVICE_TO_HOST);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
    hipSetDevice(0); hipEventRecord(mem_stop);
//
// Print output and check results
    //int output_return = output_2d_results (nx, ny,
   //     host_data_input, host_data_output);
    //if (output_return != 0) { return 0; }
//
// cufftXtFree() - Free GPU memory
    result = hipfftXtFree(device_data_input);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); return 0; }
    if (hipGetLastError()) std::cout << "Uncaught CUDA error. Line " << __LINE__ << std::endl;
//

    } else { //nGPUs == 1
    
// Demonstrate how to use CUFFT to perform 2-d FFTs using multiple GPUs
//
    hipError_t cuda_result;
    hipSetDevice(0); // just in case
// cufftCreate() - Create an empty plan
    result = hipfftCreate(&plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Create failed\n"); return 0; }
//
// cufftMakePlan2d() - Create the plan
    result = hipfftMakePlan2d (plan_input, ny, nx, HIPFFT_C2C, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed\n"); return 0; }
//
// cudaMalloc() - Malloc data on multiple GPUs
    hipfftComplex *device_data_input;
    cuda_result = hipMalloc(&device_data_input, size_of_data);
    if (cuda_result != hipSuccess) { printf ("hipMalloc failed\n"); return 0; }
    hipDeviceSynchronize(); //Sync necessary only for timing
//
// cudaMemcpy() - Copy data from host to multiple GPUs
    hipEventRecord(mem_start);
    cuda_result = hipMemcpy(device_data_input, host_data_input, size_of_data, hipMemcpyHostToDevice);
    if (cuda_result != hipSuccess) { printf ("hipMemcpy failed\n"); return 0; }
    hipDeviceSynchronize();  // sync for timing
//
// cufftExecC2C() - Execute FFT on multiple GPUs
    hipEventRecord(start);
    result = hipfftExecC2C (plan_input, device_data_input,
        device_data_input, HIPFFT_FORWARD);
    hipDeviceSynchronize(); // sync for timing
    hipEventRecord(stop);
    if (result != HIPFFT_SUCCESS) { printf ("cufftExec* failed\n"); return 0; }
//
// cudaMemcpy() - Copy data from multiple GPUs to host
    cuda_result = hipMemcpy(host_data_output, device_data_input, size_of_data, hipMemcpyDeviceToHost);
    if (cuda_result != hipSuccess) { printf ("hipMemcpy failed\n"); return 0; }
    hipEventRecord(mem_stop);
//
// Print output and check results
    //int output_return = output_2d_results (nx, ny,
   //     host_data_input, host_data_output);
    //if (output_return != 0) { return 0; }
//
// cudaFree() - Free GPU memory
    cuda_result = hipFree(device_data_input);
    if (cuda_result != hipSuccess) { printf ("hipFree failed\n"); return 0; }
    if (hipGetLastError()) std::cout << "Uncaught CUDA error. Line " << __LINE__ << std::endl;
//
    }
// report timing

    hipSetDevice(0); 
    hipEventSynchronize(stop);
    hipEventSynchronize(mem_stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << "cufft exec time: " << elapsed << " ms." << std::endl;
    std::cout << "                  " << ((float) ny) * ((float) nx)/elapsed/1024/1024 << " Mcells/ms." << std::endl;
    std::cout << "                  " <<
                 5*(((float) ny) * ((float) nx) * log2(nx) +
                    ((float) nx) * ((float) ny) * log2(ny))
                         /elapsed/1024/1024 << " GFLOPS." << std::endl;

    hipEventElapsedTime(&elapsed, mem_start, mem_stop);
    std::cout << "time including memcpy: " << elapsed << " ms." << std::endl;
// cufftDestroy() - Destroy FFT plan
    result = hipfftDestroy(plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Destroy failed: code\n"); return 0; }
    free(host_data_input); free(host_data_output);

   
// destroy timers
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(mem_start);
    hipEventDestroy(mem_stop);

    return 0;
}
