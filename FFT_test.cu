
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include <unistd.h>



int main(int argc, char** argv) {

  int sz = 1024;
  int nGPUs = 2, whichGPUs[16];
  whichGPUs[0] = 0; whichGPUs[1] = 1;
  if (argc > 1) {
    if ( ( strncmp(argv[1], "--help", 6) == 0) ||
         ( strncmp(argv[1], "-h", 2) == 0) ) {
      std::cout << "Usage:\n     FFT_test <field dimension> <num GPUs>" <<std::endl;
      return 0;
    } 
    sz = atoi(argv[1]);
  }
  int nx=sz, ny=sz;

  if (argc > 2) {
    nGPUs = atoi(argv[2]);
    if (nGPUs > 16) {
       std::cout << "More than 16 GPUs not supported" << std::endl;
       exit(1);
    }
    for (size_t qq=0;qq<nGPUs;qq++) whichGPUs[qq]=qq;
  }
  std::cout << "Transform " << ny << "x" << nx << " image "
            << "with " << nGPUs << " gpus." << std::endl;

// Timers
    hipSetDevice(0);
    hipEvent_t start, stop;
    hipEvent_t mem_start, mem_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&mem_start);
    hipEventCreate(&mem_stop);

// Demonstrate how to use CUFFT to perform 3-d FFTs using 2 GPUs
//
// cufftCreate() - Create an empty plan
    hipfftHandle plan_input; hipfftResult result; 
    result = hipfftCreate(&plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Create failed\n"); return 0; }
//
// cufftXtSetGPUs() - Define which GPUs to use
    result = hipfftXtSetGPUs (plan_input, nGPUs, whichGPUs);
    if (result != HIPFFT_SUCCESS) { printf ("*XtSetGPUs failed\n"); return 0; }
//
// Initialize FFT input data
    size_t worksize[16];
    hipfftComplex *host_data_input, *host_data_output;
    unsigned long size_of_data = sizeof(hipfftComplex) * nx * ny;
    host_data_input = (hipfftComplex *)malloc(size_of_data);
    if (host_data_input == NULL) { printf ("malloc (%ul) failed\n", size_of_data); return 0; }
    host_data_output = (hipfftComplex *)malloc(size_of_data);
    if (host_data_output == NULL) { printf ("malloc (%ul, output) failed\n", size_of_data); return 0; }
    //initialize_3d_data (nx, ny, host_data_input, host_data_output);
//
// cufftMakePlan2d() - Create the plan
    result = hipfftMakePlan2d (plan_input, ny, nx, HIPFFT_C2C, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed\n"); return 0; }
//
// cufftXtMalloc() - Malloc data on multiple GPUs
    hipLibXtDesc *device_data_input;
    result = hipfftXtMalloc (plan_input, &device_data_input,
        HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); return 0; }
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
//
// cufftXtMemcpy() - Copy data from host to multiple GPUs
    hipSetDevice(0); hipEventRecord(mem_start);
    result = hipfftXtMemcpy (plan_input, device_data_input,
        host_data_input, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
//
// cufftXtExecDescriptorC2C() - Execute FFT on multiple GPUs
    hipSetDevice(0); hipEventRecord(start);
    result = hipfftXtExecDescriptorC2C (plan_input, device_data_input,
        device_data_input, HIPFFT_FORWARD);
    for(size_t dev=0;dev<nGPUs;dev++) { hipSetDevice(whichGPUs[dev]); hipDeviceSynchronize(); }
    hipSetDevice(0); hipEventRecord(stop);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExec* failed\n"); return 0; }
//
// cufftXtMemcpy() - Copy data from multiple GPUs to host
    result = hipfftXtMemcpy (plan_input, host_data_output,
        device_data_input, HIPFFT_COPY_DEVICE_TO_HOST);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
    hipSetDevice(0); hipEventRecord(mem_stop);
//
// Print output and check results
    //int output_return = output_2d_results (nx, ny,
   //     host_data_input, host_data_output);
    //if (output_return != 0) { return 0; }
//
// cufftXtFree() - Free GPU memory
    result = hipfftXtFree(device_data_input);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); return 0; }
    if (hipGetLastError()) std::cout << "Uncaught CUDA error. Line " << __LINE__ << std::endl;
//

// report timing

    hipSetDevice(0); 
    hipEventSynchronize(stop);
    hipEventSynchronize(mem_stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << "cufft exec time: " << elapsed << " ms." << std::endl;
    hipEventElapsedTime(&elapsed, mem_start, mem_stop);
    std::cout << "including memcpy: " << elapsed << " ms." << std::endl;
// cufftDestroy() - Destroy FFT plan
    result = hipfftDestroy(plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Destroy failed: code\n"); return 0; }
    free(host_data_input); free(host_data_output);

// destroy timers
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(mem_start);
    hipEventDestroy(mem_stop);

    return 0;
}
