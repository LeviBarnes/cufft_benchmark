
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"



int main(int argc, char** argv) {

  int sz = 1024;
  if (argc > 1) sz = atoi(argv[1]);
  int nx=sz, ny=sz;
// Demonstrate how to use CUFFT to perform 3-d FFTs using 2 GPUs
//
// cufftCreate() - Create an empty plan
    hipfftHandle plan_input; hipfftResult result;
    result = hipfftCreate(&plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Create failed\n"); return 0; }
//
// cufftXtSetGPUs() - Define which GPUs to use
    int nGPUs = 2, whichGPUs[2];
    whichGPUs[0] = 0; whichGPUs[1] = 1;
    result = hipfftXtSetGPUs (plan_input, nGPUs, whichGPUs);
    if (result != HIPFFT_SUCCESS) { printf ("*XtSetGPUs failed\n"); return 0; }
//
// Initialize FFT input data
    size_t worksize[2];
    hipfftComplex *host_data_input, *host_data_output;
    int size_of_data = sizeof(hipfftComplex) * nx * ny;
    host_data_input = (hipfftComplex *)malloc(size_of_data);
    if (host_data_input == NULL) { printf ("malloc failed\n"); return 0; }
    host_data_output = (hipfftComplex *)malloc(size_of_data);
    if (host_data_output == NULL) { printf ("malloc failed\n"); return 0; }
    //initialize_3d_data (nx, ny, host_data_input, host_data_output);
//
// cufftMakePlan3d() - Create the plan
    result = hipfftMakePlan2d (plan_input, ny, nx, HIPFFT_C2C, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed\n"); return 0; }
//
// cufftXtMalloc() - Malloc data on multiple GPUs
    hipLibXtDesc *device_data_input;
    result = hipfftXtMalloc (plan_input, &device_data_input,
        HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); return 0; }
//
// cufftXtMemcpy() - Copy data from host to multiple GPUs
    result = hipfftXtMemcpy (plan_input, device_data_input,
        host_data_input, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
//
// cufftXtExecDescriptorC2C() - Execute FFT on multiple GPUs
    result = hipfftXtExecDescriptorC2C (plan_input, device_data_input,
        device_data_input, HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExec* failed\n"); return 0; }
//
// cufftXtMemcpy() - Copy data from multiple GPUs to host
    result = hipfftXtMemcpy (plan_input, host_data_output,
        device_data_input, HIPFFT_COPY_DEVICE_TO_HOST);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); return 0; }
//
// Print output and check results
    //int output_return = output_2d_results (nx, ny,
   //     host_data_input, host_data_output);
    //if (output_return != 0) { return 0; }
//
// cufftXtFree() - Free GPU memory
    result = hipfftXtFree(device_data_input);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); return 0; }
//
// cufftDestroy() - Destroy FFT plan
    result = hipfftDestroy(plan_input);
    if (result != HIPFFT_SUCCESS) { printf ("*Destroy failed: code\n"); return 0; }
    free(host_data_input); free(host_data_output);
    return 0;
}
